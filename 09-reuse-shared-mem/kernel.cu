
#include <hip/hip_runtime.h>
__inline__ __device__ float warpReduce(float value) {
    // Use XOR mode to perform butterfly reduction
    for (int i=16; i>=1; i/=2)
        value += __shfl_xor_sync(0xffffffff, value, i, 32);

    // "value" now contains the sum across all threads
    //printf("Thread %d final value = %d\n", threadIdx.x, value);
    return value;
}

__inline__ __device__ float blockReduce(float sum) {
    sum = warpReduce(sum);
    int tid = threadIdx.x;
    __shared__ float psums[16];
    if (tid % 32 == 0) {
        psums[tid / 32] = sum;
    }
    __syncthreads();

    sum = 0;
    for (int i = 0; i < blockDim.x / 32; i++) {
        sum += psums[i];
    }
    return sum;
}

extern "C" __global__
void kernel(int M, int N, float* a, float* c) {
    int m = blockIdx.x;
    int tid = threadIdx.x;

    if (m > M) {
        return;
    }

    const int ROW_SIZE = 1024*4;
    assert(N <= ROW_SIZE);

    /** Load array a into the shared memory (later reused)
     */
    __shared__ float exps[ROW_SIZE];
    for (int n = tid; n < N; n += blockDim.x) {
        if (n < N) {
            exps[n] = exp(a[m*N + n]);
        }
    }

    /** Accumulate the partial sums into the shared memory 
     */

    float sum = 0;
    for (int n = tid; n < N; n += blockDim.x) {
        if (n < N) {
            sum += exps[n];
        }
    }

    __syncthreads();

    sum = blockReduce(sum);

    for (int n = tid; n < N; n += blockDim.x) {
        if (n < N) {
            c[m*N+n] = exps[n] / sum;
        }
    }
}